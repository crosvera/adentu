#include "hip/hip_runtime.h"
/*
    Adentu: An hybrid molecular dynamic software.
    https://github.com/crosvera/adentu
    
    Copyright (C) 2013 Carlos Ríos Vera <crosvera@gmail.com>
    Universidad del Bío-Bío.

    This program is free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License
    version 3 as published by the Free Software Foundation.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include "adentu-model.h"
#include "adentu-grid.h"
#include "adentu-types.h"

extern "C" {
    #include "adentu-atom.h"
    #include "adentu-cuda.h"
    #include "adentu-types-cuda.h"
    #include "adentu-atom-cuda.h"
}


extern "C"
void adentu_atom_cuda_create_from_config (AdentuAtom *atoms, AdentuAtomConfig *conf)
{
    AdentuAtomType type = conf->type;
    int nAtoms = conf->nAtoms;
    AdentuPropRange pmass = conf->mass;
    AdentuPropRange pradii = conf->radii;

    unsigned int memsize = nAtoms * 4;

    /* allocating host side */
    adentu_real *h_pos = (adentu_real *) malloc (memsize * sizeof (adentu_real));
    adentu_real *h_vel = (adentu_real *) malloc (memsize * sizeof (adentu_real));

    adentu_real *h_velRel = NULL;
    if (type == ADENTU_ATOM_FLUID)
        h_velRel = (adentu_real *) malloc (memsize * sizeof (adentu_real));

    int *h_nCol = (int *) malloc (nAtoms * sizeof (int));
    memset (h_nCol, 0, nAtoms * sizeof (int));
    
    float *h_mass = (float *) malloc (nAtoms * sizeof (float));
    float *h_radius = (float *) malloc (nAtoms * sizeof (float));

    /* allocating device side */
    adentu_real *d_pos, *d_vel;
    ADENTU_CUDA_MALLOC (&d_pos, memsize * sizeof (adentu_real));
    ADENTU_CUDA_MALLOC (&d_vel, memsize * sizeof (adentu_real));

    adentu_real *d_velRel = NULL;
    if (type == ADENTU_ATOM_FLUID)
        ADENTU_CUDA_MALLOC (&d_velRel, memsize * sizeof (adentu_real));

    int *d_nCol;
    ADENTU_CUDA_MALLOC (&d_nCol, nAtoms * sizeof (int));
    ADENTU_CUDA_MEMSET (d_nCol, 0, nAtoms * sizeof (int));
    
    float *d_mass, *d_radius;
    ADENTU_CUDA_MALLOC (&d_mass, nAtoms * sizeof (float));
    ADENTU_CUDA_MALLOC (&d_radius, nAtoms * sizeof (float));


    for (int i = 0; i < nAtoms; ++i)
    {
        //lastTime[i] = 0;
        array4_set3v (h_pos, i, 0.0, 0.0, 0.0);
        array4_set3v (h_vel, i, 0.0, 0.0, 0.0);
        if (type == ADENTU_ATOM_FLUID)
            {
                array4_set3v (h_velRel, i, 0.0, 0.0, 0.0);
            }

        switch (pmass.rangeType) {
            case ADENTU_PROP_CONSTANT:
                h_mass[i] = pmass.from;
                break;

            case ADENTU_PROP_NORMAL:
                h_mass[i] = (float) rand() / (RAND_MAX + 1.0) * (pmass.to - pmass.from) + pmass.from;
                break;

            case ADENTU_PROP_DELTA:
                /**
                 * \todo Implement DELTA values in mass properties
                 */
                break;

            default:
                g_error ("Wrong Property Type\n");
                break;
        }

        switch (pradii.rangeType) {
            case ADENTU_PROP_CONSTANT:
                h_radius[i] = pradii.from;
                break;

            case ADENTU_PROP_NORMAL:
                h_radius[i] = (float) rand() / (RAND_MAX + 1.0) * (pradii.to - pradii.from) + pradii.from;
                break;

            case ADENTU_PROP_DELTA:
                /**
                 * \todo Implement DELTA values in radius properties
                 */
                break;

            default:
                g_error ("Wrong Property Type\n");
                break;
        }

    }

    ADENTU_CUDA_MEMCPY_H2D (d_pos, h_pos, memsize * sizeof (adentu_real));
    ADENTU_CUDA_MEMCPY_H2D (d_vel, h_vel, memsize * sizeof (adentu_real));
    ADENTU_CUDA_MEMCPY_H2D (d_mass, h_mass, nAtoms * sizeof (float));
    ADENTU_CUDA_MEMCPY_H2D (d_radius, h_radius, nAtoms * sizeof (float));
    if (type == ADENTU_ATOM_FLUID)
        ADENTU_CUDA_MEMCPY_H2D (d_velRel, h_velRel, memsize * sizeof (adentu_real));


    atoms->type = type;
    atoms->n = nAtoms;
    atoms->h_pos = h_pos;
    atoms->h_vel = h_vel;
    atoms->h_velRel = h_velRel;
    atoms->h_nCol = h_nCol;
    atoms->h_mass = h_mass;
    atoms->h_radius = h_radius;

    atoms->d_pos = d_pos;
    atoms->d_vel = d_vel;
    atoms->d_velRel = d_velRel;
    atoms->d_nCol = d_nCol;
    atoms->d_mass = d_mass;
    atoms->d_radius = d_radius;

}



/**
 ** TODO port it to CUDA 
 **/
extern "C"
void adentu_atom_cuda_set_random_vel (AdentuAtom *atoms, AdentuModel *model)
{
    vec3f _vcm = {.0, .0, .0}, vcm, v;
    double vp2 = 0.0, factor, velInit;
    const int nAtoms = atoms->n;
    adentu_real *d_vel = atoms->d_vel;
    adentu_real *h_vel = atoms->h_vel;
    float temp;


    if (atoms->type == ADENTU_ATOM_GRAIN)
        {
            velInit = (model->gTemp != 0.0) ? sqrt (3 * model->gTemp) : 0.0;
            temp = model->gTemp;
            vcm = model->vcmGrain;
        }
    else if (atoms->type == ADENTU_ATOM_FLUID)
        {
            velInit = (model->fTemp != 0.0) ? sqrt (3 * model->fTemp) : 0.0;
            temp = model->fTemp;
            vcm = model->vcmFluid;
        }

    array4Rand3f_cuda (d_vel, nAtoms);
    ADENTU_CUDA_MEMCPY_D2H (h_vel, d_vel, nAtoms * sizeof (adentu_real));

    for (int i = 0; i < nAtoms; ++i)
        {
            v = get_vec3f_from_array4f (h_vel, i);
            vecScale (v, v, velInit);
            vecAdd (_vcm, _vcm, v);
            array4_set_vec3 (h_vel, i, v);
        }
    vecScale (_vcm, _vcm, 1.0/nAtoms);

    for (int i = 0; i < nAtoms; ++i)
        {
            v = get_vec3f_from_array4f (h_vel, i);
            vecSub (v, v, _vcm);
            vp2 += vecDot (v, v);
            array4_set_vec3 (h_vel, i, v);
        }

    vp2 /= 3.0;
    factor = (vp2 != 0.0) ? sqrt (temp/vp2) : 1.0;

    for (int i = 0; i < nAtoms; ++i)
        {
            v = get_vec3f_from_array4f (h_vel, i);
            vecScale (v, v, factor);
            vecAdd (v, v, vcm);
            array4_set_vec3 (h_vel, i, v);
        }
   
    ADENTU_CUDA_MEMCPY_H2D (d_vel, h_vel, nAtoms * sizeof (adentu_real));
}



/* Next code needs to be fixed. */
/*
__global__ void adentu_atom_cuda_set_init_pos_kernel (vec3f *pos, int nAtoms, 
                                                      double *rands, double *radii, 
                                                      vec3f center, vec3f half);

extern "C"
void adentu_atom_cuda_set_random_pos (AdentuAtom *atoms, AdentuGrid *grid)
{
    int nAtoms = atoms->n;
    vec3f origin = grid->origin;
    vec3f length = grid->length;
    vec3f  half, center;
    vecScale (half, length, 0.5);
    
    center.x = origin.x + half.x;
    center.y = origin.y + half.y;
    center.z = origin.z + half.z;

    vec3f *pos, *d_pos; 
    double *d_radii, *radii;
    double *rands;
    double *d_rands;
    pos = atoms->pos;
    radii = atoms->radius;

    
    CUDA_CALL (hipMalloc ((void **)&d_pos, nAtoms * sizeof (vec3f)));
    CUDA_CALL (hipMalloc ((void **)&d_radii, nAtoms * sizeof (double)));
    CUDA_CALL (hipMalloc ((void **)&d_rands, 6 *  nAtoms * sizeof (double)));
    CUDA_CALL (hipMemcpy (d_radii, radii, nAtoms * sizeof (double), hipMemcpyHostToDevice));


    hiprandGenerator_t gen;
    hiprandCreateGenerator (&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    //hiprandSetPseudoRandomGeneratorSeed (gen, time(NULL));
    hiprandSetPseudoRandomGeneratorSeed (gen, 1234567);
    hiprandGenerateUniformDouble (gen, d_rands, nAtoms * 6);
    hiprandDestroyGenerator (gen);

    
    rands = (double *)malloc (sizeof (double) * 6 * nAtoms);
    CUDA_CALL (hipMemcpy (rands, d_rands, 6 * nAtoms * sizeof (double), hipMemcpyDeviceToHost));
    
    //for (int i =0; i < nAtoms * 6; ++i)
    //    rands[i] = drand48();
    //CUDA_CALL (hipMemcpy (d_rands, rands, 6 * nAtoms * sizeof (double), hipMemcpyHostToDevice));
    //free (rands);


    dim3 gDim (1);
    dim3 bDim (nAtoms);

    adentu_atom_cuda_set_init_pos_kernel <<<gDim, bDim>>>  (d_pos, nAtoms, 
                                                            d_rands, d_radii, 
                                                            center, half);

    CUDA_CALL (hipMemcpy (pos, d_pos, nAtoms * sizeof (vec3f), hipMemcpyDeviceToHost));

    CUDA_CALL (hipFree (d_pos));
    CUDA_CALL (hipFree (d_radii));
    CUDA_CALL (hipFree (d_rands));

}



__global__ void adentu_atom_cuda_set_init_pos_kernel (vec3f *pos, int nAtoms, 
                                                      double *rands, double *radii, 
                                                      vec3f center, vec3f half)
{

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= nAtoms)
        return ;

    vec3f p;
//    vecSet (p,  rands[idx + nAtoms * 0] - rands[idx + nAtoms * 1],
  //              rands[idx + nAtoms * 2] - rands[idx + nAtoms * 3],
    //            rands[idx + nAtoms * 4] - rands[idx + nAtoms * 5]);


    p.x =       rands[idx + nAtoms * 0] - rands[idx + nAtoms * 1];
    p.y =       rands[idx + nAtoms * 2] - rands[idx + nAtoms * 3];
    p.z =       rands[idx + nAtoms * 4] - rands[idx + nAtoms * 5];


    double r = (radii[idx]) ? radii[idx] : 1;

    p.x = center.x + (p.x * (half.x - r));
    p.y = center.y + (p.y * (half.y - r));
    p.z = center.z + (p.z * (half.z - r));

    vecSet (pos[idx], p.x, p.y, p.z);
    //pos[idx].x = p.x;
    //pos[idx].y = p.y;
    //pos[idx].z = p.z;
}

*/
