#include "hip/hip_runtime.h"
/*
    Adentu: An hybrid molecular dynamic software.
    https://github.com/crosvera/adentu
    
    Copyright (C) 2013 Carlos Ríos Vera <crosvera@gmail.com>

    This program is free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License
    version 3 as published by the Free Software Foundation.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <hip/hip_runtime.h>
#include <hiprand.h>


#include <stdio.h>
#include <glib.h>

#include "adentu-types.h"

extern "C" {
    #include "adentu-cuda.h"
}



__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                            (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                         __double_as_longlong(val +
                         __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}




extern "C"
void adentu_cuda_reset_device (void)
{
    hipDeviceReset ();
}


extern "C"
void adentu_cuda_set_grid (dim3 *gDim, dim3 *bDim, int n)
{
    if (!(n/ADENTU_CUDA_THREADS))
        gDim->x = 1;
    else
    {
        int i = n/ADENTU_CUDA_THREADS;
        int j = n % ADENTU_CUDA_THREADS;
        if (j > 0)
            gDim->x = ++i;
        else
            gDim->x = i;
    }

    gDim->y = 1;
    gDim->z = 1;

    bDim->x = ADENTU_CUDA_THREADS;
    bDim->y = 1;
    bDim->z = 1;
 
}



__global__ void adentu_cuda_integrate_atoms_kernel (adentu_real *pos,
                                                    adentu_real *vel,
                                                    double dt,
                                                    vec3f accel,
                                                    int nAtoms);

extern "C"
void adentu_cuda_integrate_atoms (AdentuAtom *atoms, 
                                  AdentuGrid *grid,
                                  const vec3f accel,
                                  const double dt)
{
    if (!atoms || !grid)
        return ;

    if (dt == 0.0)
        return ;

    int nAtoms = atoms->n;
    adentu_real *h_vel = atoms->h_vel;
    adentu_real *h_pos = atoms->h_pos;
    adentu_real *d_vel = atoms->d_vel;
    adentu_real *d_pos = atoms->d_pos;

    dim3 gDim, bDim;
    adentu_cuda_set_grid (&gDim, &bDim, nAtoms);
    adentu_cuda_integrate_atoms_kernel<<<gDim, bDim>>> (d_pos, d_vel, dt,
                                                        accel, nAtoms);

    ADENTU_CUDA_MEMCPY_D2H (h_vel, d_vel, nAtoms * 4 * sizeof (adentu_real));
    ADENTU_CUDA_MEMCPY_D2H (h_pos, d_pos, nAtoms * 4 * sizeof (adentu_real));
    
}

__global__ void adentu_cuda_integrate_atoms_kernel (adentu_real *pos,
                                                    adentu_real *vel,
                                                    double dt,
                                                    vec3f accel,
                                                    int nAtoms)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= nAtoms)
        return ;

    vec3f oldVel = get_vec3f_from_array4f (vel, idx);
    vec3f newVel = oldVel;
    vec3f newPos = get_vec3f_from_array4f (pos, idx);

    newVel.x += (accel.x * dt);
    newVel.y += (accel.y * dt);
    newVel.z += (accel.z * dt);

    newPos.x += (oldVel.x * dt + 0.5 * accel.x * dt * dt);
    newPos.y += (oldVel.y * dt + 0.5 * accel.y * dt * dt);
    newPos.z += (oldVel.z * dt + 0.5 * accel.z * dt * dt);

    array4_set_vec3 (pos, idx, newPos);
    array4_set_vec3 (vel, idx, newVel);
}
